
#include "hip/hip_runtime.h"

#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <time.h>
#include <stdio.h>

using namespace std;

#define N 2048
#define BLOCKS 256
#define THREADS 256

__global__ void gpuPiCalculate(float* localResult, hiprandState* states) {
	unsigned long id = threadIdx.x + blockDim.x * blockIdx.x;
	int V = 0;
	float x, y;

	hiprand_init(id, id, 0, &states[id]);  //initialize curand

	for (int i = 0; i < N; i++) {
		x = hiprand_uniform(&states[id]);
		y = hiprand_uniform(&states[id]);
		if (x * x + y * y < 1.0f) {
			V++;
		}
	}
	localResult[id] = 4.0f * V / (float)N;
}

float cpuPiCalculate(long n) {
	float x, y;
	long V = 0;
	for (long i = 0; i < n; i++) {
		x = rand() / (float)RAND_MAX;
		y = rand() / (float)RAND_MAX;
		V += (x * x + y * y <= 1.0f);
	}
	return 4.0f * V / n;
}

int main(int argc, char* argv[]) {
	clock_t start, stop;
	float host[BLOCKS * THREADS];
	float* dev;
	hiprandState* devStates;

	//Calc pi on GPu
	start = clock();
	hipMalloc((void**)&dev, BLOCKS * THREADS * sizeof(float));
	hipMalloc((void**)&devStates, THREADS * BLOCKS * sizeof(hiprandState));

	gpuPiCalculate <<< BLOCKS, THREADS >>> (dev, devStates);

	hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost);
	float gpuPI = 0;
	for (int i = 0; i < BLOCKS * THREADS; i++) {
		gpuPI += host[i];
	}
	gpuPI /= (BLOCKS * THREADS);
	stop = clock();

	printf("GPU PI= %f\n", gpuPI);
	printf("GPU Estimate time %f s.\n", (stop - start) / (float)CLOCKS_PER_SEC);

	//Calc pi on CPU
	start = clock();
	float cpuPI = cpuPiCalculate(BLOCKS * THREADS * N);
	stop = clock();
	printf("CPU PI= %f\n", cpuPI);
	printf("CPU Estimate time %f s.\n", (stop - start) / (float)CLOCKS_PER_SEC);

	return 0;
}